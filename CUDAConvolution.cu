#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
using namespace std;

void loadImage(string fname, float *image, char** argv){
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(fname, argv[0]);

    if (imagePath == NULL)
    {
        printf("Unable to source image file: %s\n", fname);
        exit(EXIT_FAILURE);
    }

    sdkLoadPGM(imagePath, &image, &width, &height);

    printf("Success \n");
}
int main(char** argv)
{
    float* image;
    loadImage("lena_bw.pgm", image,argv);

    return 0;
}